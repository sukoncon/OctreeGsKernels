#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <torch/torch.h>
#include <torch/extension.h>
#include <c10/util/complex.h>

#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <mma.h>

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define blockDim_x 32*2
#define blockDim_y 4

#define warp_size 32

using namespace nvcuda;

template<typename T, int pack_size>
struct GetPackType {
  using type = typename std::aligned_storage<pack_size * sizeof(T), pack_size * sizeof(T)>::type;
};

template<typename T, int pack_size>
using PackType = typename GetPackType<T, pack_size>::type;

template<typename T, int pack_size>
union Pack {

  PackType<T, pack_size> storage;
  T elem[pack_size];
};


#define colMajorIdx(rows, cols, i, j)    i + j*rows
#define rowMajorIdx(rows, cols, i, j)    i * cols + j

std::string lowercase(const std::string& inputString) {
    std::string lowerCaseInput;
    // Copying the input string into a new one for conversion:
    lowerCaseInput.resize(inputString.size());
    
    // Transform all characters of the string to lowercase 
    std::transform(
        inputString.begin(),  // First character
        inputString.end(),   // Last character
        lowerCaseInput.begin(),   // Output iterator
        ::tolower            // Unary operation applied to each char  
    );
    
    return lowerCaseInput; // Compare with desired lowercase output after conversion
}

int convertActivation(std::string activation){
  std::string loweractivation = lowercase(activation);
  if (loweractivation == "relu") return 0;
  if (loweractivation == "tanh") return 1;
  if (loweractivation == "sigmoid") return 2;
  else return -1;
}

template <typename T>
__device__ void activate(int activation,
                          T* input,
                          size_t index
                        ){
      if (activation == -1) return;
      if (activation == 0){
        input[index] = (input[index] > T(0)) ? input[index] : T(0);
      }
      else if (activation == 1){
        input[index] = tanh(input[index]);
      }
      else if (activation == 2){
        input[index] = T(1) / (T(1) + expf(-input[index]));
      }
}



__device__ half convert2half(c10::Half data){
  return data;
}

__device__ half convert2half(float data){
  return __float2half(data);
}

__device__ float convert2float(c10::Half data){
  return __half2float(data);
}

__device__ float convert2float(float data){
  return data;
}

template <typename scalar_t>
__global__ void simple2layer_wmma(
                scalar_t* input, scalar_t* weight0, scalar_t* bias0, scalar_t* weight1, scalar_t* bias1,
                scalar_t* output,
                int activation0, int activation1,
                int Mblock, int M0, int N0, int K0, int N1, int K1,
                int N0pad, int K0pad, int N1pad, int K1pad,
                int lda0Pad, int ldb0Pad, int ldb1Pad,
                int lda0, int ldb0, int ldb1){

  extern __shared__ float bufferF[];
  extern __shared__ __half bufferH[];
  __half* inSmem = bufferH;
  __half* w0Smem = bufferH + Mblock * K0pad;
  __half* w1Smem = w0Smem + K0pad * N0pad;
  __half* in1Smem = w1Smem + K1pad * N1pad;
  float* out0Smem = bufferF + (Mblock * K0pad + K0pad * N0pad + K1pad * N1pad + Mblock * N1pad) / 2;
  float* out1Smem = out0Smem + Mblock * K1pad;

  // Tile using a 2D grid
  int warpM = threadIdx.x / warpSize;
  int warpN = threadIdx.y;
  int idx = threadIdx.y * blockDim.x + threadIdx.x; // local index within a block

  // load input into shared memory
  int offset = blockIdx.x * Mblock; // row offset of this block
  for (int i = idx; i < Mblock * K0pad; i += blockDim.x * blockDim.y){
    int row = i / K0pad;
    int col = i % K0pad;
    if (col < K0 && (row + offset) < M0) inSmem[row * K0pad + col] = convert2half(input[(row + offset) * K0 + col]);
    else inSmem[row * K0pad + col] = convert2half(0.f);
  }

  // load weights into shared memory
  for (int i = idx; i < N0pad * K0pad; i += blockDim.x * blockDim.y){
    int row = i % K0pad;
    int col = i / K0pad;
    if (row < K0 & col < N0) w0Smem[row + col * K0pad] = convert2half(weight0[row + col * K0]);
    else w0Smem[row + col * K0pad] = convert2half(0.f);
  }

  for (int i = idx; i < N1pad * K1pad; i += blockDim.x * blockDim.y){
    int row = i % K1pad;
    int col = i / K1pad;
    if (row < K1 & col < N1) w1Smem[row + col * K1pad] = convert2half(weight1[row + col * K1]);
    else w1Smem[row + col * K1pad] = convert2half(0.f);
  }
  
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major>  in0_frag;
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major>  in1_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::col_major>  w0_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::col_major>  w1_frag;

  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc0_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc1_frag;
  // wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc0_frag;
  // wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> out1_frag;

  wmma::fill_fragment(acc0_frag, 0.0f);
  wmma::fill_fragment(acc1_frag, 0.0f);
  
  __syncthreads();

  // First layer
  int aRow = warpM * WMMA_M;
  int bCol = warpN * WMMA_N;
  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  for (int i = 0; i < K0pad; i += WMMA_K) {
    int aCol = i;
    int bRow = i;
    // Bounds checking
    if (bCol < N0pad & (aRow + offset) < M0) {
      // Load the inputs
      wmma::load_matrix_sync(in0_frag, inSmem + aCol + aRow * K0pad, K0pad);
      wmma::load_matrix_sync(w0_frag, w0Smem + bRow + bCol * K0pad, K0pad);

      // #pragma unroll
      // for (int t = 0; t < in0_frag.num_elements; t++) {
      //         in0_frag.x[t] =  wmma::__float_to_tf32(in0_frag.x[t]);
      // }

      // #pragma unroll
      // for (int t = 0; t < w0_frag.num_elements; t++) {
      //         w0_frag.x[t] =  wmma::__float_to_tf32(w0_frag.x[t]);
      // }

      // Perform the matrix multiplication
      wmma::mma_sync(acc0_frag, in0_frag, w0_frag, acc0_frag);
    }
  }

  if (bCol < N0pad & (aRow + offset) < M0) {
    wmma::store_matrix_sync(out0Smem  + cCol + cRow * N0pad, acc0_frag, N0pad,
                            wmma::mem_row_major);
  }
  __syncthreads();
  // bias0 & activation 0
  for (int i = idx; i < Mblock * N0pad; i += blockDim.x * blockDim.y){
    int row = i / N0pad;
    int col = i % N0pad;
    if (col < N0) {
      float tmp[1];
      tmp[0] = out0Smem[row*N0pad + col] + convert2float(bias0[col]);
      activate(activation0, tmp, 0);
      in1Smem[row*N0pad + col] = convert2half(tmp[0]);
    }else in1Smem[row*N0pad + col] = convert2half(0.f);
  }



  // __syncthreads();
  // for (int i = idx; i < Mblock * K0pad; i += blockDim.x * blockDim.y){
  //   int row = i / K0pad;
  //   int col = i % K0pad;
    
  //   if (blockIdx.x ==0){
  //     printf("idx %d, row %d, col %d, offset %d, inSmem(after) %f\n", 
  //     idx, row, col, offset, __half2float(inSmem[row*K0pad + col]));
  //   }

  // }
  // __syncthreads();
  // for (int i = idx; i < K0pad * N0pad; i += blockDim.x * blockDim.y){
  //   int row = i % K0pad;
  //   int col = i / K0pad;
    
  //   if (blockIdx.x ==0){
  //     printf("idx %d, row %d, col %d, offset %d, w0smem %f\n", 
  //     idx, row, col, offset, __half2float(w0Smem[row + K0pad * col]));
  //   }

  // }
  // __syncthreads();
  // for (int i = idx; i < K1pad * N1pad; i += blockDim.x * blockDim.y){
  //   int row = i % K1pad;
  //   int col = i / K1pad;
    
  //   // if (blockIdx.x ==0){
  //     if (row < 17 &  w1Smem[row + K1pad * col] != 1) printf("row < 17 &  w1Smem[row + K1pad * col] != 1");
  //     if (row >= 17 & w1Smem[row + K1pad * col] != 0) printf("row >= 17 & w1Smem[row + K1pad * col] != 0");
  //     // printf("idx %d, row %d, col %d, offset %d, w1smem %f\n", 
  //     // idx, row, col, offset, w1Smem[row + K1pad * col]);
  //   // }

  // // }
  // __syncthreads();
  // for (int i = idx; i < Mblock * N0pad; i += blockDim.x * blockDim.y){
  //   int row = i / N0pad;
  //   int col = i % N0pad;
    
  //   if (blockIdx.x ==0){

  //     printf("idx %d, row %d, col %d, offset %d, out0Smem %f, bias0 %f\n", 
  //     idx, row, col, offset, __half2float(in1Smem[row*N0pad + col]), bias0[col]);
  //   }

  // }
  
  __syncthreads();
  // activation0
  // Second layer
  
  for (int i = 0; i < K1pad; i += WMMA_K) {
    int aCol = i;
    int bRow = i;
    // Bounds checking
    if (bCol < N1pad & (aRow + offset) < M0) {
      // Load the inputs
      wmma::load_matrix_sync(in1_frag, in1Smem + aCol + aRow * K1pad, K1pad);
      wmma::load_matrix_sync(w1_frag, w1Smem + bRow + bCol * K1pad, K1pad); // b multiply.cu:221 if idx == 192 || idx == 256
      // #pragma unroll
      // for (int t = 0; t < in1_frag.num_elements; t++) {
      //         in1_frag.x[t] =  wmma::__float_to_tf32(in1_frag.x[t]);
      // }

      // #pragma unroll
      // for (int t = 0; t < w1_frag.num_elements; t++) {
      //         w1_frag.x[t] =  wmma::__float_to_tf32(w1_frag.x[t]);
      // }

      // Perform the matrix multiplication
      wmma::mma_sync(acc1_frag, in1_frag, w1_frag, acc1_frag);
    }
  }


  // for (int i = idx; i < Mblock * N1pad; i += blockDim.x * blockDim.y){
  //   out1Smem[i] = 0;
  // }
  // __syncthreads();
  // for (int i = idx; i < Mblock * N1pad; i += blockDim.x * blockDim.y){
  //   int row = i / N1pad;
  //   int col = i % N1pad;
    
  //   if (blockIdx.x ==0){
  //     printf("idx %d, row %d, col %d, offset %d, out1Smem(before) %f\n", 
  //     idx, row, col, offset, out1Smem[row*N1pad + col]);
  //   }

  // }

  // __syncthreads();
  
  if (cCol < N1pad & (aRow + offset) < M0) {
    wmma::store_matrix_sync(out1Smem  + cCol + cRow * N1pad, acc1_frag, N1pad,
                            wmma::mem_row_major);
  }
  __syncthreads();

  for (int i = idx; i < Mblock * N1pad; i += blockDim.x * blockDim.y){
    int row = i / N1pad;
    int col = i % N1pad;
    
    if (col<N1 & (row + offset) < M0){
      float out[1];
      out[0] = out1Smem[row*N1pad + col] + convert2float(bias1[col]); //bias 1
      activate(activation1, out, 0); //activation 1
      output[(row + offset) * N1 + col] = out[0];
    }


    // if (blockIdx.x ==0 & col<N1){
    //   printf("idx %d, row %d, col %d, offset %d, out1Smem %f, output %f\n", 
    //   idx, row, col, offset, out1Smem[row*N1pad + col], output[(row + offset) * N1 + col]);
    // }
  }

}


torch::Tensor  simple2layer(torch::Tensor& input, 
                        torch::Tensor& weight0, 
                        torch::Tensor& bias0, 
                        std::string activation0,
                        torch::Tensor& weight1, 
                        torch::Tensor& bias1, 
                        std::string& activation1) {

    int M0; int N0; int K0; int K0pad; int N0pad;
    int M1; int N1; int K1; int K1pad; int N1pad;
    int lda0Pad; int ldb0Pad; int ldb1Pad;
    int lda0; int ldb0; int ldb1;

    int act0 = convertActivation(activation0);
    int act1 = convertActivation(activation1);

    // Assertion

    if (input.is_contiguous()){
      M0 = input.size(0); K0 = input.size(1); lda0 = input.strides()[0];
      K0pad = ((K0 + WMMA_K - 1) / WMMA_K) * WMMA_K;
      lda0Pad = K0pad;
    }
    else {throw std::runtime_error("Not implemented for discontiguous INPUT tensor.");}

    if (weight0.is_contiguous()){
      N0 = weight0.size(0); ldb0 = weight0.strides()[0];
      N0pad = ((N0 + WMMA_N - 1) / WMMA_N) * WMMA_N;
      K1pad = N0pad;
      ldb0Pad = K0pad;
    }
    else {throw std::runtime_error("Not implemented for discontiguous WEIGHT0 tensor.");}

    if (weight1.is_contiguous()){
      N1 = weight1.size(0); K1 = weight1.size(1); ldb1 = weight1.strides()[0];
      N1pad = ((N1 + WMMA_N - 1) / WMMA_N) * WMMA_N;
      ldb1Pad = K1pad;
    }
    else {throw std::runtime_error("Not implemented for discontiguous WEIGHT1 tensor.");}

    torch::Tensor output = torch::empty({M0, N1}, input.options());

    dim3 gridDim;
    dim3 blockDim;

    // blockDim.x must be a multple of warpSize
    // 128x4 means we have 16 warps and a block computes a 64x64 output tile
    blockDim.x = blockDim_x;
    blockDim.y = int((max(N0, N1) + WMMA_N - 1) / WMMA_N);

    gridDim.x = std::max(int((M0 + (WMMA_M * blockDim.x / warp_size - 1)) /
                (WMMA_M * blockDim.x / warp_size)), 1);
    gridDim.y = 1; // assume N and K are small

    // store input block + weights + intermediate result in shared memory
    int Mblock = (blockDim_x/warp_size) * WMMA_M; // how many rows a block can deal with
    int smem_size = (Mblock * K0pad + K0pad * N0pad +  K1pad * N1pad + Mblock * N1pad) * sizeof(half) + // input + weights + intermediate output
                      (Mblock * N1pad + Mblock * K1pad) * sizeof(float); // intermediate output + final output

//     printf("Computing... using simple2layer_wmma kernel, blockDim.x %d, blockDim.y %d, gridDim.x %d, gridDim.y %d\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);
    AT_DISPATCH_ALL_TYPES_AND_HALF(
         input.scalar_type(), "simple2layer_wmma", ([&] {
          simple2layer_wmma<<<gridDim, blockDim, smem_size>>>
          (input.data_ptr<scalar_t>(), 
            weight0.data_ptr<scalar_t>(), 
            bias0.data_ptr<scalar_t>(), 
            weight1.data_ptr<scalar_t>(),
            bias1.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            act0, 
            act1,
            Mblock, M0, N0, K0, N1, K1, 
            N0pad,  K0pad, N1pad, K1pad, 
            lda0Pad, ldb0Pad, ldb1Pad,
            lda0,
            ldb0,
            ldb1);
    }));
    //     hipDeviceSynchronize();
//     hipError_t errAsync = hipDeviceSynchronize();
//         if (errAsync != hipSuccess)
//           printf("simple2layer Async kernel error: %s\n", hipGetErrorString(errAsync));
    hipError_t errSync  = hipGetLastError();

    if (errSync != hipSuccess)
      printf("simple2layer Sync kernel error: %s\n", hipGetErrorString(errSync));


    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    // m.def("simple_gemm", &simple_gemm, "CUDA kernel: simple_gemm");
    // m.def("smem_gemm", &smem_gemm, "CUDA kernel: smem_gemm");
    m.def("simple2layer", &simple2layer, "CUDA kernel: simple2layer");

}